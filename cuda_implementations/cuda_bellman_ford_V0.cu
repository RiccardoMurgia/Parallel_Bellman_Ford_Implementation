#include "hip/hip_runtime.h"
//
// Created by rick on 03/12/23.
//


#include "cuda_bellman_ford_V0.cuh"
#include "cuda_utilities.cuh"


__global__ void cuda_parallel_relax_edges(int *d_distances, Graph *d_graph){
    unsigned int tid = (blockIdx.x * blockDim.x) + threadIdx.x;

    if (tid < d_graph->num_edges){

        int origin = d_graph->edges[tid].origin;
        int end = d_graph->edges[tid].end;
        int weight = d_graph->edges[tid].weight;

        if (d_distances[origin] + weight < d_distances[end])
            atomicMin(&d_distances[end], d_distances[origin] + weight);
    }
}


extern "C" int cuda_bellman_ford_v0(Graph *graph, int source, int *distances, int threads_per_block){
    int num_blocks = (graph->num_edges + threads_per_block - 1) / threads_per_block;

    Graph *d_graph;
    int *d_source;
    int *d_distances;

    hipMalloc((void **) &d_graph, sizeof(Graph));
    int **gpu_adjacency_matrix_ptrs_2_free = copy_graph_2_GPU(graph, d_graph);


    hipMalloc((void **) &d_source, sizeof(int));
    hipMemcpy(d_source, &source, sizeof(int), hipMemcpyHostToDevice);

    hipMalloc((void **) &d_distances, sizeof(int) * graph->num_vertices);
    hipMemcpy(d_distances, distances, sizeof(int) * graph->num_vertices, hipMemcpyHostToDevice);

    cuda_initialize_distances<<<num_blocks, threads_per_block>>>(d_distances, d_graph, d_source);
    hipDeviceSynchronize();

    for (int i = 0; i < graph->num_vertices - 1; i++) {
        cuda_parallel_relax_edges<<<num_blocks, threads_per_block>>>(d_distances, d_graph);
        hipDeviceSynchronize();
    }

    hipMemcpy(distances, d_distances, sizeof(int) * graph->num_vertices, hipMemcpyDeviceToHost);

    freeGraph(d_graph, gpu_adjacency_matrix_ptrs_2_free, graph->num_vertices);
    hipFree(d_source);
    hipFree(d_distances);


    for (int i = 0; i < graph->num_edges; i++) {
        int origin = graph->edges[i].origin;
        int end = graph->edges[i].end;
        int weight = graph->edges[i].weight;

        if (distances[origin] + weight < distances[end])
            return 1;
    }
    return 0;
}
