#include "hip/hip_runtime.h"
//
// Created by rick on 16/05/24.
//


#include "cuda_bellman_ford_V1_1.cuh"
#include "cuda_utilities.cuh"
#include <hip/hip_runtime.h>



__device__ MinResult d_find_min_value_1(const int *array, int num_vertices, unsigned int tid){
    MinResult result;
    result.value = INT_MAX;
    result.index = -1;
    unsigned int start_index = tid * num_vertices;
    unsigned int end_index = start_index + num_vertices;

    for (unsigned int i = start_index; i < end_index; i++){
        if (array[i] < result.value) {
            result.value = array[i];
            result.index = (int)i;
        }
    }

    return result;
}


__global__ void update_distances_1(int d_group_size, int *d_dist, Graph *d_graph, int *d_new_dist, int *d_candidate_dist,
                                   volatile int *d_n_block_processed, volatile int *d_semaphore) {
    unsigned int tid = (blockIdx.x * blockDim.x) + threadIdx.x;
    unsigned int num_blocks = gridDim.x;
    int *d_tmp;


    for (int i = 0; i < d_graph->num_vertices; i++) {

        for (int j = 0; j < d_group_size; j++) {
            unsigned int g_index = tid * (d_group_size) + j;

            if (g_index < d_graph->num_vertices) {
                for (int u = 0; u < d_graph->num_vertices; u++)
                    d_candidate_dist[tid * d_graph->num_vertices + u] = d_dist[u] + d_graph->adjacency_matrix[u][tid];

                MinResult min_candidate_dist = d_find_min_value_1(d_candidate_dist, d_graph->num_vertices, tid);

                if (min_candidate_dist.value < d_dist[tid])
                    d_new_dist[tid] = min_candidate_dist.value;
                else
                    d_new_dist[tid] = d_dist[tid];

            }
            __syncthreads();

            if (threadIdx.x == 0)  // The first thread of each block notifies the end of its work
                atomicAdd((int *)d_n_block_processed, 1);

            if (tid == 0) {
                while (*d_n_block_processed != num_blocks); // The Master wait for the others block have finished

                *d_n_block_processed = 0;
                *d_semaphore = (i + 1) % 2;

            }
            while (*d_semaphore != (i + 1) % 2); // The other thread wait for the master to change the semaphore value

            d_tmp = d_dist;
            d_dist = d_new_dist;
            d_new_dist = d_tmp;

        }
    }
}


extern "C" int cuda_bellman_ford_v1_1(Graph *graph, int source, int *dist, int threads_per_block){
    int negative_cycles = 0;


    hipDeviceProp_t deviceProp{};
    hipGetDeviceProperties(&deviceProp, 0);
    int n_multi_processors = deviceProp.multiProcessorCount;
    int num_blocks = n_multi_processors + 1;
    int group_size = 0;
    int total_n_threads;

    while (num_blocks > n_multi_processors){
        group_size++;
        total_n_threads = (graph->num_vertices + group_size - 1) / group_size; // parte intera superiore numero archi / g
        num_blocks = (total_n_threads + threads_per_block - 1) / threads_per_block;
    }


    int n_block_processed = 0;
    int semaphore = 1;

    int *d_dist = nullptr;
    Graph *d_graph = nullptr;
    volatile int *d_n_block_processed = nullptr;
    volatile int *d_semaphore = nullptr;
    int *d_negative_cycles = nullptr;
    int *d_candidate_dist = nullptr;
    int *d_new_dist = nullptr;

    hipMalloc((void **) &d_dist, sizeof(int) * graph->num_vertices);
    hipMalloc((void **) &d_graph, sizeof(Graph));
    hipMalloc((void **) &d_n_block_processed, sizeof(int));
    hipMalloc((void **) &d_semaphore, sizeof(int));
    hipMalloc((void **) &d_negative_cycles, sizeof(int));
    hipMalloc((void **) &d_candidate_dist, sizeof(int) * graph->num_vertices * graph->num_vertices);
    hipMalloc((void **) &d_new_dist, sizeof(int) * graph->num_vertices);

    int **gpu_adjacency_matrix_ptrs_2_free = copy_graph_2_GPU(graph, d_graph);
    cuda_initialize_distances<<<num_blocks, threads_per_block>>>(d_dist, d_graph, source);
    hipMemcpy((void *)d_n_block_processed, (void *)&n_block_processed, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy((void *)d_semaphore, (void *)&semaphore, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_negative_cycles, &negative_cycles, sizeof(int), hipMemcpyHostToDevice);


    update_distances_1<<<num_blocks, threads_per_block>>>(group_size, d_dist, d_graph, d_new_dist,  d_candidate_dist,
                                                          d_n_block_processed, d_semaphore);


    detect_negative_cycle_1<<<num_blocks, threads_per_block>>>(d_dist, d_graph, d_negative_cycles, d_candidate_dist);
    hipMemcpy(&negative_cycles, d_negative_cycles, sizeof(int), hipMemcpyDeviceToHost);


    if(!negative_cycles)
        hipMemcpy(dist, d_dist, sizeof(int) * graph->num_vertices, hipMemcpyDeviceToHost);


    hipFree(d_new_dist);

    hipFree(d_dist);
    freeGraph(d_graph, gpu_adjacency_matrix_ptrs_2_free, graph->num_vertices);
    hipFree(d_negative_cycles);
    hipFree(d_candidate_dist);

    return negative_cycles;
}
